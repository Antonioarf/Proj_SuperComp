#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <chrono>
#include <iostream>
using namespace std;

int main(){

// Carregar os dados do arquivo de entrada na memória da GPU


// Ler os dados do arquivo de entrada
int  n_cat;
int n_filmes;
int conta = 0;
int lim;
vector<int> limites;
cin >> n_filmes >> n_cat;


thrust::device_vector<int> start_times(n_filmes);
thrust::device_vector<int> end_times(n_filmes);
thrust::device_vector<int> categories(n_filmes);
thrust::device_vector<int> L(n_cat);
while(conta<n_cat){
        cin >> lim;
        L[conta] = lim;
        conta++;
    }

conta = 0;
int i1, i2, i3;
while(conta<n_filmes){
        cin >> i1 >> i2 >> i3;
        start_times[conta] = i1;
        end_times[conta] = i2;
        categories[conta] = i3;
        conta++;
    }


// Criar a matriz de programação dinâmica
thrust::device_vector<int> dp((n_filmes+1) * (n_cat+1), 0);

// Inicializar a primeira linha da matriz com zeros
thrust::fill(dp.begin(), dp.begin() + n_cat + 1, 0);

auto before = std::chrono::high_resolution_clock::now();

// Preencher a matriz com as soluções para subproblemas menores
for (int i = 1; i <= n_filmes; i++) {
    for (int j = 1; j <= n_cat; j++) {
        // if (L[j-1] != i) {
        //     dp[i][j] = dp[i-1][j];
        // } else {
        //     int last = i - 1;
        //     while (last >= 0 && end_times[last] > start_times[i-1]) {
        //         last--;
        //     }
    // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
    int max_count = 0;
    for (int k = 0; k < i; k++) {
      if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(n_cat+1)) + j-1] + 1 <= L[j-1]) {
        max_count = max(max_count, dp[(k*(n_cat+1)) + j-1] + 1);
      } else {
        max_count = max(max_count, dp[(k*(n_cat+1)) + j]);
      }
    }
    dp[(i*(n_cat+1)) + j] = max_count;
        }
  }
//}

// Encontrar o número máximo de filmes que podem ser assistidos
int max_count = 0;
for (int j = 1; j <= n_cat; j++) {
  max_count = max(max_count, dp[(n_filmes*(n_cat+1)) + j]);
}


    ///// FIM DO TIMER
    auto after = chrono::high_resolution_clock::now();
    auto delta = chrono::duration_cast<chrono::nanoseconds>(after-before).count();

    cout << "Time: " << delta << "\n";
    cout << "N_filmes " << n_filmes << "\n";
    cout << "N_cat " << n_cat << "\n";
    cout << "TAMANHO " << max_count << "\n";
}


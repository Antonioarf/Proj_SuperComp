#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>
#include <iostream>
using namespace std;



struct MaxFilmesFunctor {
    int N;
    int M;
    const int* categories;
    const int* end_times;
    const int* start_times;
    const int* dp;
    const int* L;

    MaxFilmesFunctor(int N_, int M_, const int* categories_, const int* end_times_, const int* start_times_, const int* dp_, const int* L_) :
        N(N_), M(M_), categories(categories_), end_times(end_times_), start_times(start_times_), dp(dp_), L(L_) {}

    __host__ __device__
    int operator()(const thrust::tuple<int, int>& indices) {
        int i = thrust::get<0>(indices) + 1;
        int j = thrust::get<1>(indices) + 1;

        // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
        int max_count = 0;
        for (int k = 0; k < i; k++) {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= L[j-1]) {
                max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
            } else {
                max_count = max(max_count, dp[(k*(M+1)) + j]);
            }
        }

        return max_count;
    }
};


int main(){

// Carregar os dados do arquivo de entrada na memória da GPU


// Ler os dados do arquivo de entrada
int  n_cat;
int n_filmes;
int conta = 0;
int lim;
vector<int> limites;
cin >> n_filmes >> n_cat;


thrust::host_vector<int> start_times(n_filmes);
thrust::host_vector<int> end_times(n_filmes);
thrust::host_vector<int> categories(n_filmes);
thrust::host_vector<int> L(n_cat);
while(conta<n_cat){
        cin >> lim;
        L[conta] = lim;
        conta++;
    }

conta = 0;
int i1, i2, i3;
while(conta<n_filmes){
        cin >> i1 >> i2 >> i3;
        start_times[conta] = i1;
        end_times[conta] = i2;
        categories[conta] = i3;
        conta++;
    }




auto before = std::chrono::high_resolution_clock::now();

thrust::host_vector<int> dp{(n_filmes+1) * (n_cat+1), 0};
thrust::device_vector<int> dp_gpu = dp;

thrust::device_vector<int> categories_gpu = categories;
thrust::device_vector<int> end_times_gpu = end_times;
thrust::device_vector<int> start_times_gpu = start_times;
thrust::device_vector<int> L_gpu = L;

// Criar a sequência de índices
thrust::counting_iterator<int> first(0);
thrust::counting_iterator<int> last = first + N * M;




thrust::device_vector<int> result(n_filmes * n_cat);
thrust::transform(first, last, result.begin(), MaxFilmesFunctor(n_filmes,n_cat, thrust::raw_pointer_cast(categories_gpu.data()), thrust::raw_pointer_cast(end_times_gpu.data()), thrust::raw_pointer_cast(start_times_gpu.data()), thrust::raw_pointer_cast(dp_gpu.data()), thrust::raw_pointer_cast(L_gpu.data())));

// Copiar o resultado de volta para a CPU
thrust::host_vector<int> result_cpu = result;


int max_count = *thrust::max_element(result_cpu.begin(), result_cpu.end());

    

    ///// FIM DO TIMER
auto after = chrono::high_resolution_clock::now();
auto delta = chrono::duration_cast<chrono::nanoseconds>(after-before).count();

cout << "Time: " << delta << "\n";
cout << "N_filmes " << n_filmes << "\n";
cout << "N_cat " << n_cat << "\n";
cout << "TAMANHO " << max_count << "\n";
}


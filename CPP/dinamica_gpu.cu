#include "hip/hip_runtime.h"
%%writefile dinamica2.cu
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>

#include <chrono>
#include <iostream>

using namespace std;

struct UpdateFunctor
{
    int* categories;
    int* end_times;
    int* start_times;
    int* dp;
    int* L;
    int M;

    __host__ __device__
    UpdateFunctor(int* _categories, int* _end_times, int* _start_times, int* _dp, int* _L, int _M)
        : categories(_categories), end_times(_end_times), start_times(_start_times), dp(_dp), L(_L), M(_M)
    {
    }

    __host__ __device__
    int operator()(int i) const
    {
        int max_count = 0;
        int j = threadIdx.x + blockIdx.x * blockDim.x;

        for (int k = 0; k < i; k++)
        {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k * (M + 1)) + j - 1] + 1 <= L[j - 1])
            {
                max_count = max(max_count, dp[(k * (M + 1)) + j - 1] + 1);
            }
            else
            {
                max_count = max(max_count, dp[(k * (M + 1)) + j]);
            }
        }

        dp[(i * (M + 1)) + j] = max_count;
        return max_count;
    }
};
int main()
{
  auto before = std::chrono::high_resolution_clock::now();

    int N; // Number of elements in categories and end_times
    int M;  // Number of columns in dp array
    
    std::cin >> N >> M;

    // Input data
    int categories[N] ;
    int end_times[N] ;
    int start_times[N] ;
    int dp[(N + 1) * (M + 1)];
    int L[M];

    int  conta = 0;
    int  lim   =0;
    while(conta<M){
            std::cin >> lim;
            L[conta] = lim;
            conta++;
        }

    conta = 0;
    int i1, i2, i3;
    while(conta<N){
            std::cin >> i1 >> i2 >> i3;
            start_times[conta] = i1;
            end_times[conta] = i2;
            categories[conta] = i3;
            conta++;
        }

    for (int i =0;i<N;i++){
        cout << categories[i] << " " << end_times[i] << " " << start_times[i] << endl;
    }

    // Transfer input data to device
    thrust::device_vector<int> d_categories(categories, categories + N);
    thrust::device_vector<int> d_end_times(end_times, end_times + N);
    thrust::device_vector<int> d_start_times(start_times, start_times + N);
    thrust::device_vector<int> d_dp(dp, dp + (N + 1) * (M + 1));
    thrust::device_vector<int> d_L(L, L + M);


    thrust::fill(d_dp.begin(), d_dp.begin() + M + 1, 1);
    
    int numElements = ((N + 1) * (M + 1));
    thrust::counting_iterator<int> first(10);
    thrust::counting_iterator<int> last = first + numElements;

    // // Launch the kernel and update the dp array
     UpdateFunctor functor(thrust::raw_pointer_cast(d_categories.data()),
                           thrust::raw_pointer_cast(d_end_times.data()),
                           thrust::raw_pointer_cast(d_start_times.data()),
                           thrust::raw_pointer_cast(d_dp.data()),
                           thrust::raw_pointer_cast(d_L.data()),
                           M);

     thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(numElements),
                       d_dp.begin(), functor);

    // Transfer result back to host
    thrust::copy(d_dp.begin(), d_dp.end(), dp);

    // Print the updated dp array
    int max_count = 0;
    for (int j = 1; j <= M; j++) {
      max_count = max(max_count, dp[(N*(M+1)) + j]);
      //std::cout<< ((N*(M+1)) + j)<< std::endl;
    }
    auto after = chrono::high_resolution_clock::now();
    auto delta = chrono::duration_cast<chrono::nanoseconds>(after-before).count();
    std::cout << delta<< " , " << N<< " , " << M<< " , "<< max_count << "\n";
    return 0;
}

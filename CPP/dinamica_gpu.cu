#include "hip/hip_runtime.h"
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <chrono>
#include <iostream>

using namespace std;



struct MaxFilmesFunctor {
    int n_filmes; // Number of filmes (movies)
    int n_cat; // Number of categories
    int* categories;
    int* end_times;
    int* start_times;
    int* dp;
    int* L;

    MaxFilmesFunctor(int n_filmes, int n_cat, int* categories, int* end_times, int* start_times, int* dp, int* L) : 
    n_filmes(n_filmes), n_cat(n_cat), categories(categories), end_times(end_times), start_times(start_times), dp(dp), L(L) {}

    __device__
    int operator()() {


        for (int i = 1; i <= n_filmes; i++) {
        for (int j = 1; j <= n_cat; j++) {
            // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
            int max_count = 0;
            for (int k = 0; k < i; k++) {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(n_cat+1)) + j-1] + 1 <= L[j-1]) {
                max_count = max(max_count, dp[(k*(n_cat+1)) + j-1] + 1);
            } else {
                max_count = max(max_count, dp[(k*(n_cat+1)) + j]);
            }
            }
            dp[(i*(n_cat+1)) + j] = max_count;
        }
        }
    }
};


int main(){

// Carregar os dados do arquivo de entrada na memória da GPU


// Ler os dados do arquivo de entrada
int  n_cat;
int n_filmes;
int conta = 0;
int lim;
vector<int> limites;
cin >> n_filmes >> n_cat;


thrust::host_vector<int> start_times(n_filmes);
thrust::host_vector<int> end_times(n_filmes);
thrust::host_vector<int> categories(n_filmes);
thrust::host_vector<int> L(n_cat);
while(conta<n_cat){
        cin >> lim;
        L[conta] = lim;
        conta++;
    }

conta = 0;
int i1, i2, i3;
while(conta<n_filmes){
        cin >> i1 >> i2 >> i3;
        start_times[conta] = i1;
        end_times[conta] = i2;
        categories[conta] = (i3-1);
        conta++;
    }




auto before = std::chrono::high_resolution_clock::now();

thrust::host_vector<int> dp{(n_filmes+1) * (n_cat+1), 0};
thrust::device_vector<int> dp_gpu = dp;

thrust::device_vector<int> categories_gpu = categories;
thrust::device_vector<int> end_times_gpu = end_times;
thrust::device_vector<int> start_times_gpu = start_times;
thrust::device_vector<int> L_gpu = L;

// Criar a sequência de índices
thrust::counting_iterator<int> first(0);
thrust::counting_iterator<int> last = first + n_filmes * n_cat;




thrust::device_vector<int> result(n_filmes * n_cat, 0);

thrust::transform(thrust::device,
                  thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(n_filmes * n_cat),
                  result.begin(),
                  MaxFilmesFunctor(n_filmes, n_cat,
                                    raw_pointer_cast(categories_gpu.data()),
                                    raw_pointer_cast(end_times_gpu.data()),
                                    raw_pointer_cast(start_times_gpu.data()),
                                    raw_pointer_cast(dp_gpu.data()),
                                    raw_pointer_cast(L_gpu.data())));
    // Encontrar o número máximo de filmes que podem ser assistidos
int max_count = 0;
// for (int j = 1; j <= n_cat; j++) {
//   max_count = max(max_count, dp[(n_filmes*(n_cat+1)) + j]);
// }
    

    ///// FIM DO TIMER
auto after = chrono::high_resolution_clock::now();
auto delta = chrono::duration_cast<chrono::nanoseconds>(after-before).count();

cout << delta<< " , " << n_filmes<< " , " << n_cat<< " , "<< max_count<< "\n";
}

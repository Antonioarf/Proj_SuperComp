#include "hip/hip_runtime.h"
// %%writefile dinamica3.cu
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>

#include <chrono>
#include <iostream>

using namespace std;
struct UpdateFunctor
{
    int* categories;
    int* end_times;
    int* start_times;
    int* dp;
    int* L;
    int M;

    __host__ __device__
    UpdateFunctor(int* _categories, int* _end_times, int* _start_times, int* _dp, int* _L, int _M)
        : categories(_categories), end_times(_end_times), start_times(_start_times), dp(_dp), L(_L), M(_M)
    {
    }

    __host__ __device__
    int operator()(int i) const
    {
        int max_count = 0;
        int j = threadIdx.x + blockIdx.x * blockDim.x;

        for (int k = 0; k < i; k++)
        {
            if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k * (M + 1)) + j - 1] + 1 <= L[j - 1])
            {
                max_count = max(max_count, dp[(k * (M + 1)) + j - 1] + 1);
            }
            else
            {
                max_count = max(max_count, dp[(k * (M + 1)) + j]);
            }
        }

        dp[(i * (M + 1)) + j] = max_count;
        return max_count;
    }
};
int main()
{
    auto before = std::chrono::high_resolution_clock::now();

    int N; // Number of elements in categories and end_times
    int M;  // Number of columns in dp array
    
    std::cin >> N >> M;

    // Input data
    thrust::host_vector<int> categories(N) ;
    thrust::host_vector<int> end_times(N) ;
    thrust::host_vector<int> start_times(N) ;
    thrust::host_vector<int> dp((N + 1) * (M + 1));
    thrust::host_vector<int> L(M);

    int  conta = 0;
    int  lim   =0;
    while(conta<M){
            std::cin >> lim;
            L[conta] = lim;
            conta++;
        }

    conta = 0;
    int i1, i2, i3;
    while(conta<N){
            std::cin >> i1 >> i2 >> i3;
            start_times[conta] = i1;
            end_times[conta] = i2;
            categories[conta] = i3;
            conta++;
        }

    // Transfer input data to device
    thrust::device_vector<int> d_categories = categories;
    thrust::device_vector<int> d_end_times = end_times;
    thrust::device_vector<int> d_start_times = start_times;
    thrust::device_vector<int> d_dp = dp;
    thrust::device_vector<int> d_L = L;


    thrust::fill(d_dp.begin(), d_dp.begin() + M + 1, 1);
    
    int numElements = ((N + 1) * (M + 1));
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + numElements;

    // // Launch the kernel and update the dp array
     UpdateFunctor functor(thrust::raw_pointer_cast(d_categories.data()),
                           thrust::raw_pointer_cast(d_end_times.data()),
                           thrust::raw_pointer_cast(d_start_times.data()),
                           thrust::raw_pointer_cast(d_dp.data()),
                           thrust::raw_pointer_cast(d_L.data()),
                           M);

     thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(numElements),
                       d_dp.begin(), functor);

    // Transfer result back to host
    dp=d_dp;

    // Print the updated dp array


    // Print the updated dp array
    int max_count = 0;
    for (int j = 1; j <= M; j++) {
      max_count = max(max_count, dp[(N*(M+1)) + j]);
      //std::cout<< ((N*(M+1)) + j)<< std::endl;
    }

    auto after = chrono::high_resolution_clock::now();
    auto delta = chrono::duration_cast<chrono::nanoseconds>(after-before).count();
    std::cout << delta<< " , " << N<< " , " << M<< " , "<< max_count << "\n";
    return 0;
}
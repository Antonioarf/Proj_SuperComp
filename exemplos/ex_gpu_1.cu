#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <bit>
#include <bitset>
#include <cstdint>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

struct Filme{
    int inicio;
    int fim;
    int categoria;
};


void preenche_horarios(int &horarios_disponiveis, int inicio, int fim){
    for (int i = 0; i < 24; i++){
        if (i >= inicio && i < fim){
            horarios_disponiveis  |= (1 << i);
        }
        else if (inicio > fim && (i >= inicio || i < fim)){
            horarios_disponiveis |= (1 << i);
        }
        else if(inicio == fim){
            horarios_disponiveis |= (1 << i);
        }
    }
}

struct busca_exaustiva_gpu 
{  
    int qtd_filmes;
    int qtd_categorias;
    int* filmes_por_categoria;
    int* horario_filmes;
    int* categoria_filmes;
    busca_exaustiva_gpu(int qtd_filmes_, int qtd_categorias_, int* filmes_por_categoria_, int* horario_filmes_, int* categoria_filmes_) : 
    qtd_filmes(qtd_filmes_), qtd_categorias(qtd_categorias_), filmes_por_categoria(filmes_por_categoria_), horario_filmes(horario_filmes_), categoria_filmes(categoria_filmes_) {}
    __host__ __device__
    int operator()(const int& config) {
        int horarios_disponiveis = 0;
        int filmes_por_categoria_aux[20];
        for (int i = 0; i < qtd_categorias; i++){
            filmes_por_categoria_aux[i] = *(filmes_por_categoria+i);
        }
        int num_filmes = 0;
        for (int i = 0; i < qtd_filmes; i++){
            if (config & (1 << i)){
                if (filmes_por_categoria_aux[categoria_filmes[i]-1] > 0){
                    int horario_analisado = horarios_disponiveis & *(horario_filmes + i);
                    if (((horario_analisado) != 0)) return false;
                    filmes_por_categoria_aux[categoria_filmes[i]-1]--;
                    horarios_disponiveis = horarios_disponiveis | *(horario_filmes + i);
                    num_filmes++;
                }
            }
        
        }

        return num_filmes;
    }
};



// int main(){
//     int qtd_filmes, qtd_categorias;
//     cin >> qtd_filmes >> qtd_categorias;

//     thrust::host_vector<int> filmes_por_categoria(qtd_categorias, 0);
//     Filme filme_vazio = {0, 0, 0};
//     vector<Filme> vetor_filmes (qtd_filmes, filme_vazio);

//     for (int i = 0; i < qtd_categorias; i++){
//         cin >> filmes_por_categoria[i];
//     }

//     for (int i = 0; i < qtd_filmes; i++){
//         Filme filme;
//         cin >> filme.inicio >> filme.fim >> filme.categoria;
//         if (filme.inicio == 0) filme.inicio = 24;
//         if (filme.fim == 0) filme.fim = 24;
//         if (filme.inicio < 0 || filme.fim < 0) continue;

//         vetor_filmes[i] = filme;
//     }

//     thrust::host_vector<int> categoria_filmes(qtd_filmes);
//     thrust::host_vector<int> horarios_filmes_cpu(qtd_filmes); 


//     for (int i = 0; i < qtd_filmes; i++){
//         horarios_filmes_cpu[i] = 0;
//         preenche_horarios(horarios_filmes_cpu[i], vetor_filmes[i].inicio-1, vetor_filmes[i].fim-1);
//         categoria_filmes[i] = vetor_filmes[i].categoria;
//     }

    thrust::device_vector<int> config_vector_gpu(pow(2, qtd_filmes));

    thrust::sequence(config_vector_gpu.begin(), config_vector_gpu.end());

    thrust::device_vector<int> categoria_filmes_gpu(categoria_filmes);
    thrust::device_vector<int> horarios_filmes_gpu(horarios_filmes_cpu);
    thrust::device_vector<int> filmes_por_categoria_gpu(filmes_por_categoria);

    thrust::transform(config_vector_gpu.begin(), config_vector_gpu.end(), config_vector_gpu.begin(), busca_exaustiva_gpu(qtd_filmes, qtd_categorias, raw_pointer_cast(filmes_por_categoria_gpu.data()), raw_pointer_cast(horarios_filmes_gpu.data()), raw_pointer_cast(categoria_filmes_gpu.data())));
    
    thrust::host_vector<int> config_vector_cpu_final = config_vector_gpu;
    
    int max_count = 0;
    int iters = pow(2, qtd_filmes);
    for (int i = 0; i < iters; i++){
        if (config_vector_cpu_final[i] > max_count){
            max_count = config_vector_cpu_final[i];
        }
    }

    cout << "Foram vistos " << max_count << " filmes." << endl;
}




// g++ -Wl,-z,stack-size=4194304 exaustiva.cpp -o exaustiva
//  g++ -Wl,-z,stack-size=6000000000 -fopenmp exaustiva.cpp -o exaustiva
// user@monstrinho:~/ProjetoSupercomp$ ./exaustiva 
// nvcc -arch=sm_70 -rdc=true -o exaustiva_thrust exaustiva_thrust.cu












#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <iostream>



int main() {
    // Dados de entrada
    int N = 3;
    int M = 2;
    thrust::host_vector<int> categories{1, 2, 2};
    thrust::host_vector<int> end_times{2, 4, 5};
    thrust::host_vector<int> start_times{1, 3, 4};
    thrust::host_vector<int> dp{(N+1) * (M+1), 0};
    thrust::host_vector<int> L{2, 1};

    // Transferir os dados para a GPU
    thrust::device_vector<int> categories_gpu = categories;
    thrust::device_vector<int> end_times_gpu = end_times;
    thrust::device_vector<int> start_times_gpu = start_times;
    thrust::device_vector<int> dp_gpu = dp;
    thrust::device_vector<int> L_gpu = L;

    // Criar a sequência de índices
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + N * M;

    // Calcular o número máximo de filmes para cada combinação de i e j
    thrust::device_vector<int> result(N * M);
    thrust::transform(first, last, result.begin(), MaxFilmesFunctor(N, M, thrust::raw_pointer_cast(categories_gpu.data()), thrust::raw_pointer_cast(end_times_gpu.data()), thrust::raw_pointer_cast(start_times_gpu.data()), thrust::raw_pointer_cast(dp_gpu.data()), thrust::raw_pointer_cast(L_gpu.data())));

    // Copiar o resultado de volta para a CPU
    thrust::host_vector<int> result_cpu = result;

    // Encontrar o valor máximo
    int max_count = *thrust::max_element(result_cpu.begin(), result_cpu.end());

    // Imprimir o resultado
    std::cout << "Número máximo de filmes que podem ser assistidos: " << max_count << std::endl;

    return 0;
}
